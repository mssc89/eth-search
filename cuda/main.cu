#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstddef>
#include <limits>
#include <fstream>
#include <sstream>
#include <cstdint>
#include <cinttypes>
#include <stdio.h>
#include <stdint.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include <cuco/extent.cuh>
#include <cuco/detail/hash_functions/xxhash.cuh>
#include <cuco/static_set.cuh>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <hip/hip_cooperative_groups.h>

#include "./secp256k1/inc_vendor.h"
#include "./secp256k1/inc_types.h"
#include "./secp256k1/inc_ecc_secp256k1.h"
#include "./secp256k1/inc_ecc_secp256k1.cl"

#include "./keccak/keccak256.h"
#include "./keccak/keccak256.cl"

// little endian to big endian
__device__ u32 le_to_be(u32 x)
{
    return ((x & 0xFF) << 24) | (((x >> 8) & 0xFF) << 16) | (((x >> 16) & 0xFF) << 8) | ((x >> 24) & 0xFF);
}

// get keccak256 hash in big endian
__device__ void keccak256_get_hash_be(u32* r, const u8* msg, const u32 len)
{
    u64 state[25] = {};
    keccak256_update_state(state, (u8*)msg, len);

    r[0] = le_to_be((u32)(state[1] >> 32));
    r[1] = le_to_be((u32)state[2]);
    r[2] = le_to_be((u32)(state[2] >> 32));
    r[3] = le_to_be((u32)state[3]);
    r[4] = le_to_be((u32)(state[3] >> 32));
}

// convert u32 to hexadecimal
__device__ void u32_to_hex(std::byte* result, u32* k, size_t len){
    for (size_t i = 0; i < len; ++i) {
        uint32_t value = (uint32_t)k[i];
        for (int j = 0; j < 8; ++j) {
            int nibble = (value >> (28 - j * 4)) & 0xF;
            result[i * 8 + j] = std::byte((nibble < 10) ? ('0' + nibble) : ('a' + nibble - 10));
        }
    }
}

// secp256k1 g point
__constant__ secp256k1_t g = {
    0x16f81798, 0x59f2815b, 0x2dce28d9, 0x029bfcdb,
    0xce870b07, 0x55a06295, 0xf9dcbbac, 0x79be667e,
    0xfb10d4b8, 0x9c47d08f, 0xa6855419, 0xfd17b448,
    0x0e1108a8, 0x5da4fbfc, 0x26a3c465, 0x483ada77,
    0x04ef2777, 0x63b82f6f, 0x597aabe6, 0x02e84bb7,
    0xf1eef757, 0xa25b0403, 0xd95c3b9a, 0xb7c52588,
    0xbce036f9, 0x8601f113, 0x836f99b0, 0xb531c845,
    0xf89d5229, 0x49344f85, 0x9258c310, 0xf9308a01,
    0x84b8e672, 0x6cb9fd75, 0x34c2231b, 0x6500a999,
    0x2a37f356, 0x0fe337e6, 0x632de814, 0x388f7b0f,
    0x7b4715bd, 0x93460289, 0xcb3ddce4, 0x9aff5666,
    0xd5c80ca9, 0xf01cc819, 0x9cd217eb, 0xc77084f0,
    0xb240efe4, 0xcba8d569, 0xdc619ab7, 0xe88b84bd,
    0x0a5c5128, 0x55b4a725, 0x1a072093, 0x2f8bde4d,
    0xa6ac62d6, 0xdca87d3a, 0xab0d6840, 0xf788271b,
    0xa6c9c426, 0xd4dba9dd, 0x36e5e3d6, 0xd8ac2226,
    0x59539959, 0x235782c4, 0x54f297bf, 0x0877d8e4,
    0x59363bd9, 0x2b245622, 0xc91a1c29, 0x2753ddd9,
    0xcac4f9bc, 0xe92bdded, 0x0330e39c, 0x3d419b7e,
    0xf2ea7a0e, 0xa398f365, 0x6e5db4ea, 0x5cbdf064,
    0x087264da, 0xa5082628, 0x13fde7b5, 0xa813d0b8,
    0x861a54db, 0xa3178d6d, 0xba255960, 0x6aebca40,
    0xf78d9755, 0x5af7d9d6, 0xec02184a, 0x57ec2f47,
    0x79e5ab24, 0x5ce87292, 0x45daa69f, 0x951435bf
};

// xxHash64 hasher
__shared__ cuco::detail::XXHash_64<char> hasher;

// main kernel
template <typename SetRef>
__global__ void genEthAddressSeq(SetRef set, u32 *host_count)
{
    // generated address
    u32 r[5];

    // private key
    u32 k[8];

    // secp256k1 result coordinates
    u32 x[8];
    u32 y[8];

    // keccak256 input
    u32 w[16];

    // get first element of private key
    k[0] = blockIdx.x * blockDim.x + threadIdx.x;

    // set the rest of private key
    for(int i=1; i < 8; i++){
        k[i] = host_count[i-1];
    }

    // run secp256k1
    point_mul_xy(x, y, k, &g);

    // convert coordinates to big endian
    for (int i = 0; i < 8; ++i) {
        w[i] = le_to_be(x[7 - i]);
        w[i + 8] = le_to_be(y[7 - i]);
    }

    // run keccak256
    keccak256_get_hash_be(r, (u8 *)w, 64);

    // convert to hexadecimal
    std::byte result[40];
    u32_to_hex(result, r, 5);

    // calculate xxHash
    std::uint64_t outhash = hasher.compute_hash(result, cuco::extent<std::size_t, 40>{});

    // search for address in set
    auto tile = cooperative_groups::tiled_partition<SetRef::cg_size>(cooperative_groups::this_thread_block());

    tile.sync();

    if (set.contains(tile, outhash) && tile.thread_rank() == 0) {
        printf("!!! Found collision !!!:\n%u, %u, %u, %u, %u, %u, %u, %u | %u, %u, %u, %u, %u\nAddress: %s\nxxHash (dec): %lu\n", (unsigned int)k[7], (unsigned int)k[6], (unsigned int)k[5], (unsigned int)k[4], (unsigned int)k[3], (unsigned int)k[2], (unsigned int)k[1], (unsigned int)k[0], (unsigned int)r[0], (unsigned int)r[1], (unsigned int)r[2], (unsigned int)r[3], (unsigned int)r[4], result, outhash);
    }
}

void incrementWithRollover(u32* number) {
    for (int i = 0; i < 7; i++) {
        number[i]++;

        if (number[i] != 0) {
            break;
        }
    }
}

int main(void) {
    std::cout << "Loading known addresses..." << std::endl;

    using Key = uint64_t;

    // empty slots are represented by reserved "sentinel" values. These values should be selected such that they never occur in input data
    Key constexpr empty_key_sentinel = 0;
  
    // number of keys to be inserted
    std::size_t constexpr num_keys = 277699467;
  
    // compute capacity based on a 50% load factor
    auto constexpr load_factor = 0.5;
    std::size_t const capacity = std::ceil(num_keys / load_factor);
  
    // constructs a set with at least `capacity` slots using 0 as the empty keys sentinel.
    cuco::static_set<Key> set{capacity, cuco::empty_key{empty_key_sentinel}};

    thrust::host_vector<Key> keys_host(num_keys);
  
    std::ifstream inputFile("known_addresses_hashed.csv");
    if (!inputFile.is_open()) {
        std::cerr << "Error opening file!" << std::endl;
        return 1;
    }
  
    uint64_t currentNumber;
    double lastProgress = 0.0;
  
    // load keys and output progress
    for (std::size_t i = 0; i < num_keys; ++i) {
      inputFile >> currentNumber;
      keys_host[i] = currentNumber;

      double progress = static_cast<double>(i) / num_keys * 100.0;

      if (progress - lastProgress >= 1.0) {
        std::cout << "Loading progress: " << progress << "%" << std::endl;
        lastProgress = progress;
      }
    }
  
    inputFile.close();
  
    thrust::device_vector<Key> keys = keys_host;

    // insert all keys into the hash set
    set.insert(keys.begin(), keys.end());

    std::cout << "Done loading known addresses" << std::endl << "First 5 addresses (xxhash64):" << std::endl;

    //print first 5 addresses
    for (std::size_t i = 0; i < 5; ++i) {
        std::cout << keys[i] << std::endl;
    }

    // number of "blocks" to be generated
    // each block is 4294967296 addresses
    int numBlocks = 10;

    hipSetDevice(0);

    // starter key (0)
    u32 hostArray[7] = {0,0,0,0,0,0,0};
    u32 *deviceArray;

    // allocate memory on the GPU
    hipMalloc(&deviceArray, 7 * sizeof(u32));

    // generation loop
    for (int i = 0; i < numBlocks; i++) {
        // create events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // copy starter key to GPU
        hipMemcpy(deviceArray, hostArray, 7 * sizeof(u32), hipMemcpyHostToDevice);
            
        // run kernel
        genEthAddressSeq<<<8388608, 512>>>(set.ref(cuco::contains), deviceArray);
        
        // synchronize and check for errors
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();

        if(err != hipSuccess)
        {
            std::cout << "CUDA Error: " << hipGetErrorString(err);
        }

        // copy result (last key) back to CPU
        hipMemcpy(hostArray, deviceArray, 7 * sizeof(u32), hipMemcpyDeviceToHost);

        // output the result (last key)
        std::cout << "Result: ";
        for (int j = 0; j < 7; ++j) {
            std::cout << hostArray[j] << " ";
        }
        std::cout << std::endl;

        // increment the key
        incrementWithRollover(hostArray);

        // stop timing
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // calculate time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Calculate runs per second
        unsigned long long int totalRuns = 0xFFFFFFFF;
        unsigned long long int runsPerSecond = totalRuns / (milliseconds / 1000.0f);
        printf("miliseconds: %f\n", milliseconds);
        printf("number of runs: %llu\n", totalRuns);
        printf("Number of runs per second: %.2f Mh/s\n", runsPerSecond / 1e6);
        printf("Number of runs per second: %llu h/s\n", runsPerSecond);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    return 0;
}

